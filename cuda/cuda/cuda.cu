#include "hip/hip_runtime.h"
#include "cuda.cuh"

// cuda procedure
//
// gpu memory allocation
// copy cpu->gpu
// calculation- / kernel phase
// copy gpu->cpu
// gpu memory free

__global__ void test()
{

	return;
}

// rgb->Ycbcr
__global__ void dev_convertColorSpace(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{
	int channels = 3;
	int globalThreadId = (blockIdx.x * channels) * blockDim.x + (threadIdx.x*channels);

	// grid-stride loop
	for(int dataElement = globalThreadId; dataElement < (dataSize-(channels)); dataElement= dataElement+(gridDim.x * blockDim.x)*(channels))
	{
		unsigned char r = dev_data[dataElement + 0];
		unsigned char g = dev_data[dataElement +1];
		unsigned char b = dev_data[dataElement +2];

		dev_dataResult[dataElement + 0] = 16 + (((r << 6) + (r << 1) + (g << 7) + g + (b << 4) + (b << 3) + b) >> 8); // Y
		dev_dataResult[dataElement + 1] = 128 + (((r << 7) - (r << 4) - ((g << 6) + (g << 5) - (g << 1)) - ((b << 4) + (b << 1))) >> 8); // Cb
		dev_dataResult[dataElement + 2] = 128 + ((-((r << 5) + (r << 2) + (r << 1)) - ((g << 6) + (g << 3) + (g << 1)) + (b << 7) - (b << 4)) >> 8); // Cr
	}
}


//__global__ void dev_applyGaussian(unsigned char* dev_data, unsigned char* dev_dataResult, double* filter, int dataSize, int imageHeight, int imageWidth, int filterHeight)
//{
//	int blockId = blockIdx.x + blockIdx.y * gridDim.x+ gridDim.x * gridDim.y * blockIdx.z;
//	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)+ (threadIdx.z * (blockDim.x * blockDim.y))+ (threadIdx.y * blockDim.x) + threadIdx.x;
//	int currentIndex = threadId;
//
//	int imageYSource = currentIndex / imageWidth;
//	int imageXSource = currentIndex % imageWidth;
//
//	int cuttedAwayTotal = filterHeight / 2;
//
//	if (imageYSource <  cuttedAwayTotal || imageYSource >(imageHeight -1 -cuttedAwayTotal) || imageXSource < cuttedAwayTotal || imageXSource >(imageWidth -1 -cuttedAwayTotal))
//	{
//		return;
//	}
//
//	int newImageHeight = imageHeight - filterHeight+1;
//	int newImageWidth = imageWidth - filterHeight+1;
//
//	//height
//	for (int h = 0; h < filterHeight; h++)
//	{
//		//width
//		for (int w = 0; w < filterHeight; w++)
//		{
//			double tmp = filter[h * filterHeight + w] * dev_data[(imageYSource + h) * imageWidth + (imageXSource + w)];
//
//			dev_dataResult[(imageYSource- cuttedAwayTotal) * newImageWidth + (imageXSource- cuttedAwayTotal)] += tmp;
//		}
//	}
//
//
//	//// max = 7000000 with block- and grid-dim = 1
//	//for (int i = 0; i < 1; i++) {
//	//	dev_dataResult[0] = 1;
//	//}
//
//}

__global__ void dev_applyGaussianALL(unsigned char* dev_data, unsigned char* dev_dataResult, double* filter, int dataSize, int imageHeight, int imageWidth, int filterHeight)
{

	int channels = 3;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	int currentIndex = threadId;

	int imageYSource = currentIndex / (channels * imageWidth);
	int imageXSource = currentIndex % (channels * imageWidth);

	int cuttedAway = filterHeight / 2;

	int newImageHeight = imageHeight - filterHeight + 1;
	int newImageWidth = imageWidth - filterHeight + 1;

	int currentChannel = currentIndex % channels;


	if (!(imageYSource < cuttedAway || imageYSource >(imageHeight - 1 - cuttedAway) || imageXSource < cuttedAway * channels || imageXSource >(imageWidth * channels - 1 - cuttedAway * channels)))
	{
		////height
		for (int h = 0; h < filterHeight; h++)
		{
			//width
			for (int w = 0; w < filterHeight; w++)
			{
				double tmp = filter[h * filterHeight + w] * dev_data[((imageYSource + h) * (channels * imageWidth) + (imageXSource + (channels * w)))];
				dev_dataResult[((imageYSource - cuttedAway) * (channels * newImageWidth) + (imageXSource - channels * cuttedAway))] += tmp;

			}
		}
	}

	//// max = 7000000 with block- and grid-dim = 1
//for (int i = 0; i < 1; i++) {
//	dev_dataResult[0] = 1;
//}

}

unsigned char * convertRGBToYCBCR(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	unsigned char* dataResult = (unsigned char*) malloc(sizeof(unsigned char) * dataSize);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_convertColorSpace <<< gridDims, blockDims >>> (dev_data, dev_dataResult, dataSize);

	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);

	hipFree(&dev_data);
	hipFree(&dev_dataResult);
	hipDeviceReset();

	return dataResult;
}


//unsigned char* gaussianOneChannel(unsigned char * data, int dataSize, dim3 gridDims, dim3 blockDims, double* filter, int imageHeight, int imageWidth, int filterHeight)
//{
//
//	int channels = 3;
//	int newImageWidth = imageWidth - filterHeight + 1;
//	int newImageHeight = imageHeight - filterHeight + 1;
//	int dataSizeResultImage = newImageWidth * newImageHeight;
//	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSizeResultImage);
//
//	unsigned char* dev_data;
//	unsigned char* dev_dataResult;
//	double* dev_filter;
//
//	int tmp = dataSize / channels;
//
//
//	hipMalloc(&dev_filter, sizeof(double) * filterHeight * filterHeight);
//	hipMemcpy(dev_filter, filter, sizeof(double) * filterHeight * filterHeight, hipMemcpyHostToDevice);
//
//
//
//	hipMalloc(&dev_data, sizeof(unsigned char) * tmp);
//	hipError_t error = hipGetLastError();
//	if (error != hipSuccess)
//	{
//		printf("CUDA error0: %s\n", hipGetErrorString(error));
//		exit(-1);
//	}
//	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSizeResultImage);
//
//	error = hipGetLastError();
//	if (error != hipSuccess)
//	{
//		printf("CUDA error1: %s\n", hipGetErrorString(error));
//		exit(-1);
//	}
//	hipMemcpy(dev_data, data, sizeof(unsigned char) * (dataSize / channels), hipMemcpyHostToDevice);
//	error = hipGetLastError();
//	if (error != hipSuccess)
//	{
//		printf("CUDA error2: %s\n", hipGetErrorString(error));
//		exit(-1);
//	}
//
//	dev_applyGaussian << < gridDims, blockDims >> > (dev_data, dev_dataResult, dev_filter, dataSize, imageHeight, imageWidth, filterHeight);
//
//	error = hipGetLastError();
//	if (error != hipSuccess)
//	{
//		printf("CUDA error3: %s\n", hipGetErrorString(error));
//		exit(-1);
//	}
//
//
//	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSizeResultImage, hipMemcpyDeviceToHost);
//
//
//	//hipFree(&dev_data);
//	//hipFree(&dev_dataResult);
//	//hipFree(&dev_filter);
//	//hipDeviceReset();
//
//	return dataResult;
//}

unsigned char* gaussianAllChannel(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims, double* filter, int imageHeight, int imageWidth, int filterHeight)
{

	int channels = 3;
	int newImageWidth = channels*(imageWidth - filterHeight + 1);
	int newImageHeight = channels*(imageHeight - filterHeight + 1);
	int dataSizeResultImage = newImageWidth * newImageHeight;
	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSizeResultImage);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	double* dev_filter;



	hipMalloc(&dev_filter, sizeof(double) * filterHeight * filterHeight);
	hipMemcpy(dev_filter, filter, sizeof(double) * filterHeight * filterHeight, hipMemcpyHostToDevice);



	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error0: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSizeResultImage); //dataSizeResultImage

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error1: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error2: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	dev_applyGaussianALL << < gridDims, blockDims >> > (dev_data, dev_dataResult, dev_filter, dataSize, imageHeight, imageWidth, filterHeight);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error3: %s\n", hipGetErrorString(error));
		exit(-1);
	}


	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSizeResultImage, hipMemcpyDeviceToHost); //

	//for (int i = 0; i < newImageHeight * newImageWidth; i++)
	//{
	//	cout << (int)dataResult[i] << " ";
	//}

	//hipDeviceSynchronize();

	//hipFree(&dev_data);
	//hipFree(&dev_dataResult);
	//hipFree(&dev_filter);
	//hipDeviceReset();

	return dataResult;
}




double* createGaussianFilter(int width, int height, double sigma)
{
    double r, s = 2.0 * sigma * sigma; 

    

	double PI = 3.1415;

	double** kernel = (double**) malloc(height * sizeof(double *));

	for (int i = 0; i < height; i++) {
		kernel[i] = (double*) malloc(sizeof(double) * width);
	}

	double sum = 0.0;


	for (int x = -height/2; x <= height/2; x++) { 
        for (int y = -height/2; y <= height/2; y++) { 
            r = sqrt(x * x + y * y); 
			kernel[x + height/2][y + height/2] = (exp(-(r * r) / s)) / (PI * s);
            sum += kernel[x + height/2][y + height/2];
        } 
    } 

  
    // normalising the Kernel 
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < height; ++j) {
			kernel[i][j] /= sum;
        }
	} 

	double* kernelFlat = (double*)malloc(height * height * sizeof(double));

	for (int h = 0; h < height; h++){
		for (int w = 0; w < height; w++){
			kernelFlat[h * height + w] = kernel[h][w]; // y*width+width_pos
		}
	}

	return kernelFlat;
}

unsigned char* applyGaussianFilter(unsigned char* data, int dataSize, dim3 gridDims,
									dim3 blockDims, const int channelsPara, int imageHeight,
									int imageWidth, int filterHeight, double sigma)
{
	double* filter = createGaussianFilter(filterHeight, filterHeight, sigma);
	unsigned char* resultData = gaussianAllChannel(data, dataSize, gridDims, blockDims,
													filter, imageHeight, imageWidth, filterHeight);
	return resultData;
}
