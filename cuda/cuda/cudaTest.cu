#include "hip/hip_runtime.h"
#include "cudaTest.cuh"

// cuda procedure
//
// gpu memory allocation
// copy cpu->gpu
// calculation- / kernel phase
// copy gpu->cpu
// gpu memory free

__global__ void test()
{

	return;
}

// rgb->Ycbcr
__global__ void dev_convertColorSpace(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{
	int offset = 0;
	int channels = 3;
	for(int dataElement = 0; dataElement < (dataSize*channels); dataElement+channels)
	{
		unsigned char* r = (dev_dataResult + offset+0)
		unsigned char* g = *(dev_dataResult + offset+1)
		unsigned char* b = *(dev_dataResult + offset+2)
		*r = 16+ (((*r << 6) + (*r << 1) + (*g << 7) + *g + (*b << 4) + (*b << 3) + *b) >> 8); // Y
		*g= 128 + ((-((*r<<5)+(*r<<2)+(*r<<1))-((*g<<6)+(*g<<3)+(*g<<1))+(*b<<7)-(*b<<4))>>8);; // Cb
		*b = 128 + (((*r<<7)-(*r<<4)-((*g<<6)+(*g<<5)-(*g<<1))-((*b<<4)+(*b<<1)))>>8);; // Cr
	}
}

__global__ void dev_applyGaussian(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{


	*dev_dataResult = 1;



    double stdv = 1.0;
    double r, s = 2.0 * stdv * stdv;  // Assigning standard deviation to 1.0
    double sum = 0.0;   // Initialization of sun for normalization
    for (int x = -2; x <= 2; x++) // Loop to generate 5x5 kernel
    {
        for(int y = -2; y <= 2; y++)
        {
            r = sqrt(x*x + y*y);
            gk[x + 2][y + 2] = (exp(-(r*r)/s))/(M_PI * s);
            sum += gk[x + 2][y + 2];
        }
    }

    for(int i = 0; i < 5; ++i) // Loop to normalize the kernel
        for(int j = 0; j < 5; ++j)
            gk[i][j] /= sum;

}


void doSmth()
{
	std::cout << "Hi from doSmth" << std::endl;;
	test << < 1, 1 >> > ();
	return;
}

unsigned char * convertBGRToYCBCR(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	unsigned char* dataResult = (unsigned char*) malloc(sizeof(unsigned char) * dataSize);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_convertColorSpace <<< gridDims, blockDims >>> (dev_data, dev_dataResult, dataSize);

	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);

	hipFree(&dev_data);
	hipFree(&dev_dataResult);



	return dataResult;
}


unsigned char* applyGaussianFilter(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSize); // ver�ndert sich die output-size?

	unsigned char* dev_data;
	unsigned char* dev_dataResult;

	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize); // ver�ndert sich die output-size?

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_applyGaussian << < gridDims, blockDims >> > (dev_data, dev_dataResult, dataSize);

	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);

	hipFree(&dev_data);
	hipFree(&dev_dataResult);

	return data;
}





void cudaMain(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	doSmth();
	convertBGRToYCBCR(data, dataSize, gridDims, blockDims);
	//applyGaussianFilter(data, dataSize, gridDims, blockDims);
}

