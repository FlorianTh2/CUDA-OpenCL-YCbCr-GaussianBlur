#include "hip/hip_runtime.h"
#include "cudaTest.cuh"

// cuda procedure
//
// gpu memory allocation
// copy cpu->gpu
// calculation- / kernel phase
// copy gpu->cpu
// gpu memory free

__global__ void test()
{

	return;
}

// rgb->Ycbcr
__global__ void dev_convertColorSpace(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{
	int channels = 3;
	//dataSize /= 3;
	// blockid * wieVieleBl�ckeGesamt + (threadID*channels)
	int globalThreadId = (blockIdx.x * channels) * blockDim.x + (threadIdx.x*channels);

	// grid-stride loop
	for(int dataElement = globalThreadId; dataElement < (dataSize-(channels)); dataElement= dataElement+(gridDim.x * blockDim.x)*(channels))
	{
		unsigned char r = dev_data[dataElement + 0];
		unsigned char g = dev_data[dataElement +1];
		unsigned char b = dev_data[dataElement +2];
		//*r = 16+ (((*r << 6) + (*r << 1) + (*g << 7) + *g + (*b << 4) + (*b << 3) + *b) >> 8); // Y
		//*g= 128 + ((-((*r<<5)+(*r<<2)+(*r<<1))-((*g<<6)+(*g<<3)+(*g<<1))+(*b<<7)-(*b<<4))>>8); // Cb
		//*b = 128 + (((*r<<7)-(*r<<4)-((*g<<6)+(*g<<5)-(*g<<1))-((*b<<4)+(*b<<1)))>>8); // Cr
		//*(dev_dataResult + dataElement + 0) = *r;
		//*(dev_dataResult + dataElement + 1) = *g;
		//*(dev_dataResult + dataElement + 2) = *b;

		dev_dataResult[dataElement + 0] = 16 + (((r << 6) + (r << 1) + (g << 7) + g + (b << 4) + (b << 3) + b) >> 8); // Y
		dev_dataResult[dataElement + 1] = 128 + ((-((r << 5) + (r << 2) + (r << 1)) - ((g << 6) + (g << 3) + (g << 1)) + (b << 7) - (b << 4)) >> 8); // Cb
		dev_dataResult[dataElement + 2] = 128 + (((r << 7) - (r << 4) - ((g << 6) + (g << 5) - (g << 1)) - ((b << 4) + (b << 1))) >> 8); // Cr
	}
}


//// d_paddedImage: speicherallokierung mit ->  paddedIWidth * paddedIHeight * sizeof(float)
//const T* d_f,
//
//// paddedIWidth = iWidth + 2 * hFilterSize // hFilterSize = filterSize / 2 = eig radius des kernels
//const unsigned int paddedW,
//
//
//// paddedIHeight = iHeight + 2 * hFilterSize // hFilterSize = filterSize / 2 = eig radius des kernels
//const unsigned int paddedH,
//
//// radius of filter, sp�ter zusammengetragen mit filterSize=(2S+1)�(2S+1)(2S+1)�(2S+1).
//const int S,
//
//// d_filteringResult speicherallokierung mit // iWidth * iHeight * sizeof(float)
//T* d_h,
//
//// image widthe
//const unsigned int W,
//
////image height
//const unsigned int H )


__global__ void dev_applyGaussian(unsigned char* dev_data, unsigned char* dev_dataResult, double** filter, int dataSize, int imageHeight, int imageWidth, int filterHeight)
{

	int newImageHeight = imageHeight - filterHeight + 1;
	int newImageWidth = imageWidth - filterHeight + 1;


		for (int i = 0; i < newImageWidth; i++) {
			for (int j = 0; j < newImageHeight; j++) {
				for (int h = i; h < i + filterHeight; h++) {
					for (int w = j; w < j + filterHeight; w++) {
		//				dev_dataResult[i* imageWidth + j] = (unsigned char) (dev_dataResult[i * imageWidth + j] + filter[h - i][w - j] * dev_data[h * imageWidth + w]);
		//				dev_dataResult[0] = 1;
					}
				}
			}
		}

		dev_dataResult[0] = 1;

}




























void doSmth()
{
	std::cout << "Hi from doSmth" << std::endl;
	test << < 1, 1 >> > ();
	return;
}



unsigned char * convertRGBToYCBCR(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	cout << dataSize << endl;
	unsigned char* dataResult = (unsigned char*) malloc(sizeof(unsigned char) * dataSize);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_convertColorSpace <<< gridDims, blockDims >>> (dev_data, dev_dataResult, dataSize);


	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);


	hipFree(&dev_data);
	hipFree(&dev_dataResult);



	return dataResult;
}



unsigned char* gaussianOneChannel(unsigned char * data, int dataSize, dim3 gridDims, dim3 blockDims, double** filter, int imageHeight, int imageWidth, int filterHeight)
{


	int newImageWidth = imageWidth - filterHeight + 1;
	int newImageHeight = imageHeight - filterHeight + 1;
	int dataSizeResultImage = newImageWidth * newImageHeight;
	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSizeResultImage);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;

	int tmp = dataSize / 3;




	hipMalloc(&dev_data, sizeof(unsigned char) * tmp);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error0: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSizeResultImage);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error1: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipMemcpy(dev_data, data, sizeof(unsigned char) * (dataSize / 3), hipMemcpyHostToDevice);
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error2: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	dev_applyGaussian << < gridDims, blockDims >> > (dev_data, dev_dataResult, filter, dataSize, imageHeight, imageWidth, filterHeight);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error3: %s\n", hipGetErrorString(error));
		exit(-1);
	}


	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSizeResultImage, hipMemcpyDeviceToHost);
	//hipFree(&dev_data);
	//hipFree(&dev_dataResult);

































































	//irgendwie wird der Datensatz nicht richtig initialisiert oder Null-pointer oder kA jedenfalls kann ich im kernel keine 1 setzen, die auf host ersichtlich wird
	// geht doch, aber besonders bei der innersten loop kommt manchmal 1 durch, oft aber 205, das deuted meiner Meinung nach auf Speicherallokierungsproblem hin
	// bez�glich eines Parameters von dev_applyGaussian


	for (size_t i = 0; i < dataSizeResultImage; i++)
	{
		cout << (int) dataResult[i] << " ";
	}

	return dataResult;
}

double** createGaussianFilter(int width, int height, double sigma)
{
	double PI = 3.1415;

	double** kernel = (double**) malloc(height * sizeof(double *));

	for (int i = 0; i < height; i++) {
		kernel[i] = (double*) malloc(sizeof(double) * width);
	}


	double sum = 0.0;
	int a, b;

	for (a = 0; a < height; a++) {

		for (b = 0; b < width; b++) {
			double result = exp(-(a * a + b * b) / (2 * sigma * sigma)) / (2 * PI * sigma * sigma);
			cout << "result: " << result << endl;
			kernel[a][b] = result;
			sum += kernel[a][b];
		}
	}

	for (a = 0; a < height; a++) {

		for (b = 0; b < width; b++) {
			kernel[a][b] /= sum;
		}
	}

	cout << "sum: " << sum << endl;

	//for (a = 0; a < width; a++)
	//{
	//	for (b = 0; b < height; b++)
	//	{
	//		cout << (double) kernel[a][b] << " ";
	//	}
	//	cout << endl;
	//}

	return kernel;
}

// data: BGR-Sequence of the input channels of data
unsigned char** applyGaussianFilter(unsigned char** data, int dataSize, dim3 gridDims, dim3 blockDims, const int channelsPara, int imageHeight, int imageWidth, int filterHeight, double sigma)
{


	cout << dataSize << endl;
	const int channels = channelsPara;

	const int sizeOfOneColorChannel = dataSize / 3;

	unsigned char** resultChannels1 = (unsigned char**)malloc(channels * sizeof(unsigned char*));

	double** filter = createGaussianFilter(filterHeight, filterHeight, sigma);

	int newImageWidth = imageWidth - filterHeight + 1;
	int newImageHeight = imageHeight - filterHeight + 1;
	int dataSizeResultImage = newImageWidth * newImageHeight;

	for (int i = 0; i < channels; i++) {
		resultChannels1[i] = (unsigned char*) malloc(sizeof(unsigned char) * dataSizeResultImage);
		//resultChannels1[i] = gaussianOneChannel(data[i], dataSize, gridDims, blockDims, filter, imageHeight, imageWidth, filterHeight);
	}
	resultChannels1[0] = gaussianOneChannel(data[0], dataSize, gridDims, blockDims, filter, imageHeight, imageWidth, filterHeight);


	for (size_t i = 0; i < dataSize/3; i++)
	{
		//cout << (int) data[0] << " ";
	}
	hipDeviceReset();

	return resultChannels1;
}




void cudaMain(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	doSmth();
	convertRGBToYCBCR(data, dataSize, gridDims, blockDims);
	//applyGaussianFilter(data, dataSize, gridDims, blockDims);
}

