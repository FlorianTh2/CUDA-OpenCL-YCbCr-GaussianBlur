#include "hip/hip_runtime.h"
#include "cudaTest.cuh"

// cuda procedure
//
// gpu memory allocation
// copy cpu->gpu
// calculation- / kernel phase
// copy gpu->cpu
// gpu memory free

__global__ void test()
{

	return;
}

// rgb->Ycbcr
__global__ void dev_convertColorSpace(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{
	int channels = 3;
	//dataSize /= 3;
	// blockid * wieVieleBl�ckeGesamt + (threadID*channels)
	int globalThreadId = (blockIdx.x * channels) * blockDim.x + (threadIdx.x*channels);

	// grid-stride loop
	for(int dataElement = globalThreadId; dataElement < (dataSize-(channels)); dataElement= dataElement+(gridDim.x * blockDim.x)*(channels))
	{
		unsigned char r = dev_data[dataElement + 0];
		unsigned char g = dev_data[dataElement +1];
		unsigned char b = dev_data[dataElement +2];
		//*r = 16+ (((*r << 6) + (*r << 1) + (*g << 7) + *g + (*b << 4) + (*b << 3) + *b) >> 8); // Y
		//*g= 128 + ((-((*r<<5)+(*r<<2)+(*r<<1))-((*g<<6)+(*g<<3)+(*g<<1))+(*b<<7)-(*b<<4))>>8); // Cb
		//*b = 128 + (((*r<<7)-(*r<<4)-((*g<<6)+(*g<<5)-(*g<<1))-((*b<<4)+(*b<<1)))>>8); // Cr
		//*(dev_dataResult + dataElement + 0) = *r;
		//*(dev_dataResult + dataElement + 1) = *g;
		//*(dev_dataResult + dataElement + 2) = *b;

		dev_dataResult[dataElement + 0] = 16 + (((r << 6) + (r << 1) + (g << 7) + g + (b << 4) + (b << 3) + b) >> 8); // Y
		dev_dataResult[dataElement + 1] = 128 + ((-((r << 5) + (r << 2) + (r << 1)) - ((g << 6) + (g << 3) + (g << 1)) + (b << 7) - (b << 4)) >> 8); // Cb
		dev_dataResult[dataElement + 2] = 128 + (((r << 7) - (r << 4) - ((g << 6) + (g << 5) - (g << 1)) - ((b << 4) + (b << 1))) >> 8); // Cr
	}
}


__global__ void dev_applyGaussian(unsigned char* dev_data, unsigned char* dev_dataResult, double* filter, int dataSize, int imageHeight, int imageWidth, int filterHeight)
{

	int imageYSource = blockIdx.x / imageWidth;
	int imageXSource = blockIdx.x % imageWidth;

	int cuttedAway = (filterHeight/2)

	if (imageYSource <  cuttedAway && imageYSource > (imageHeight - cuttedAway -1) && imageXSource <  cuttedAway && imageXSource > (imageWidth - cuttedAway -1))
		return;


	int newImageHeight = imageHeight - filterHeight + 1;
	int newImageWidth = imageWidth - filterHeight + 1;

	int imageYResult = newImageWidth / newImageWidth;
	int imageXResult = newImageWidth % newImageWidth;





		//for (int h = 0; h < filterHeight; h++)
		//{
		//	for (int w = 0; w < filterHeight; w++)
		//	{
		//		double tmp = filter[h * filterHeight + w] * dev_data[(imageYResult + h - filterHeight/2) * newImageWidth + (imageXResult + w - filterHeight / 2)];
		//
		//		dev_dataResult[imageYResult * newImageWidth + imageXResult] += filter[4 * filterHeight + 4] * tmp* 100; // (dev_dataResult[imageYResult * newImageWidth + imageXResult] + tmp * 1000000);
		//	}
		//}

		for (int h = 0; h < filterHeight; h++)
		{
			for (int w = 0; w < filterHeight; w++)
			{
				double tmp = filter[h * filterHeight + w] * dev_data[(imageYSource + h) * imageWidth + (imageXSource + w)];

				dev_dataResult[(imageYSource-cuttedAway) * newImageWidth + (imageXSource- cuttedAway)] += tmp * 100;
			}
		}


		//dev_dataResult[0] =  100 * filter[0];
		//dev_dataResult[1] =  dev_data[17051];
		//dev_dataResult[2] = filter[4 * filterHeight + 4] * 100000;


		

	//// max = 7000000 with block- and grid-dim = 1
	//for (int i = 0; i < 1; i++) {
	//	dev_dataResult[0] = 1;
	//}


}




























void doSmth()
{
	std::cout << "Hi from doSmth" << std::endl;
	test << < 1, 1 >> > ();
	return;
}



unsigned char * convertRGBToYCBCR(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	cout << dataSize << endl;
	unsigned char* dataResult = (unsigned char*) malloc(sizeof(unsigned char) * dataSize);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);



	dev_convertColorSpace <<< gridDims, blockDims >>> (dev_data, dev_dataResult, dataSize);


	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);


	hipFree(&dev_data);
	hipFree(&dev_dataResult);



	return dataResult;
}



unsigned char* gaussianOneChannel(unsigned char * data, int dataSize, dim3 gridDims, dim3 blockDims, double* filter, int imageHeight, int imageWidth, int filterHeight)
{


	int newImageWidth = imageWidth - filterHeight + 1;
	int newImageHeight = imageHeight - filterHeight + 1;
	int dataSizeResultImage = newImageWidth * newImageHeight;
	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSizeResultImage);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	double* dev_filter;

	int tmp = dataSize / 3;


	hipMalloc(&dev_filter, sizeof(double) * filterHeight * filterHeight);
	hipMemcpy(dev_filter, filter, sizeof(double) * filterHeight * filterHeight, hipMemcpyHostToDevice);



	hipMalloc(&dev_data, sizeof(unsigned char) * tmp);
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error0: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSizeResultImage);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error1: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipMemcpy(dev_data, data, sizeof(unsigned char) * (dataSize / 3), hipMemcpyHostToDevice);
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error2: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	dev_applyGaussian << < gridDims, blockDims >> > (dev_data, dev_dataResult, dev_filter, dataSize, imageHeight, imageWidth, filterHeight);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error3: %s\n", hipGetErrorString(error));
		exit(-1);
	}


	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSizeResultImage, hipMemcpyDeviceToHost);




	for (size_t i = 0; i < dataSizeResultImage; i++)
	{
		//cout << (int) dataResult[i] << " ";



		//if (data[i] != 0 && data[i] != 255) {
		//	cout << "This index of element not equals 0 and not 255: " << i << endl;
		//}
		//cout << (int) data[i] << " ";
	}

	hipFree(&dev_data);
	hipFree(&dev_dataResult);
	hipFree(&dev_filter);
	hipDeviceReset();

	return dataResult;
}

double* createGaussianFilter(int width, int height, double sigma)
{
        double r, s = 2.0 * sigma * sigma; 

    

	double PI = 3.1415;

	double** GKernel = (double**) malloc(height * sizeof(double *));

	for (int i = 0; i < height; i++) {
		GKernel[i] = (double*) malloc(sizeof(double) * width);
	}

	double sum = 0.0;


for (int x = -height/2; x <= height/2; x++) { 
        for (int y = -height/2; y <= height/2; y++) { 
            r = sqrt(x * x + y * y); 
            GKernel[x + height/2][y + height/2] = (exp(-(r * r) / s)) / (M_PI * s); 
            sum += GKernel[x + height/2][y + height/2]; 
        } 
    } 
  
    // normalising the Kernel 
    for (int i = 0; i < height; ++i) 
    {
        for (int j = 0; j < height; ++j)
        {
            GKernel[i][j] /= sum; 
        }
	} 


    for (int i = 0; i < height; ++i) 
    {
        for (int j = 0; j < height; ++j)
        {
            cout << GKernel[i][j] << " ";
        }
        cout << endl;
	} 






	cout << "sum: " << sum << endl;


	double* kernelFlat = (double*)malloc(height * height * sizeof(double));

	for (int h = 0; h < height; h++)
	{
		for (int w = 0; w < height; w++)
		{
			// y*width+width_pos
			kernelFlat[h * height + w] = GKernel[h][w];
		}
	}


	return kernelFlat;
}

// data: BGR-Sequence of the input channels of data
unsigned char** applyGaussianFilter(unsigned char** data, int dataSize, dim3 gridDims, dim3 blockDims, const int channelsPara, int imageHeight, int imageWidth, int filterHeight, double sigma)
{


	cout << dataSize << endl;
	const int channels = channelsPara;

	const int sizeOfOneColorChannel = dataSize / 3;

	unsigned char** resultChannels1 = (unsigned char**)malloc(channels * sizeof(unsigned char*));

	double* filter = createGaussianFilter(filterHeight, filterHeight, sigma);

	int newImageWidth = imageWidth - filterHeight + 1;
	int newImageHeight = imageHeight - filterHeight + 1;
	int dataSizeResultImage = newImageWidth * newImageHeight;

	for (int i = 0; i < channels; i++) {
		resultChannels1[i] = (unsigned char*) malloc(sizeof(unsigned char) * dataSizeResultImage);
		resultChannels1[i] = gaussianOneChannel(data[i], dataSize, gridDims, blockDims, filter, imageHeight, imageWidth, filterHeight);
	}
	//resultChannels1[0] = gaussianOneChannel(data[0], dataSize, gridDims, blockDims, filter, imageHeight, imageWidth, filterHeight);


	for (size_t i = 0; i < dataSize/3; i++)
	{
		//cout << (int) data[0] << " ";
	}
	hipDeviceReset();

	return resultChannels1;
}




void cudaMain(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	doSmth();
	convertRGBToYCBCR(data, dataSize, gridDims, blockDims);
	//applyGaussianFilter(data, dataSize, gridDims, blockDims);
}




	//width
	//for (int i = 0; i < newImageWidth; i++) {
	//	//height
	//	for (int j = 0; j < newImageHeight; j++) {
	//		//filterWidth
	//		//used "filterHeight" for height and width because its assumed that filter is symmetric
	//		for (int h = i; h < (i + filterHeight); h++) {
	//			//filterHeight
	//			for (int w = j; w < (j + filterHeight); w++) {
	//				//dev_dataResult[i* imageWidth + j] = (unsigned char) (dev_dataResult[i * imageWidth + j] + filter[h - i][w - j] * dev_data[h * imageWidth + w]);
	//				//dev_dataResult[0] = 1;
	//			}
	//		}
	//	}
	//}

