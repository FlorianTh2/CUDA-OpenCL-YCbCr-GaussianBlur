#include "hip/hip_runtime.h"
#include "cudaTest.cuh"

// cuda procedure
//
// gpu memory allocation
// copy cpu->gpu
// calculation- / kernel phase
// copy gpu->cpu
// gpu memory free

__global__ void test()
{

	return;
}

__global__ void dev_convertColorSpace(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{


	*dev_dataResult = 1;

}

__global__ void dev_applyGaussian(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{


	*dev_dataResult = 1;

}


void doSmth()
{
	std::cout << "Hi from doSmth" << std::endl;;
	test << < 1, 1 >> > ();
	return;
}

unsigned char * convertBGRToYCBCR(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	unsigned char* dataResult = (unsigned char*) malloc(sizeof(unsigned char) * dataSize);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_convertColorSpace <<< gridDims, blockDims >>> (dev_data, dev_dataResult, dataSize);

	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);

	hipFree(&dev_data);
	hipFree(&dev_dataResult);



	return dataResult;
}


unsigned char* applyGaussianFilter(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSize); // ver�ndert sich die output-size?

	unsigned char* dev_data;
	unsigned char* dev_dataResult;

	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize); // ver�ndert sich die output-size?

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_applyGaussian << < gridDims, blockDims >> > (dev_data, dev_dataResult, dataSize);

	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);

	hipFree(&dev_data);
	hipFree(&dev_dataResult);

	return data;
}





void cudaMain(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	doSmth();
	convertBGRToYCBCR(data, dataSize, gridDims, blockDims);
	//applyGaussianFilter(data, dataSize, gridDims, blockDims);
}

