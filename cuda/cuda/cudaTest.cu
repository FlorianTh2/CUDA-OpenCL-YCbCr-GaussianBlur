#include "hip/hip_runtime.h"
#include "cudaTest.cuh"

// cuda procedure
//
// gpu memory allocation
// copy cpu->gpu
// calculation- / kernel phase
// copy gpu->cpu
// gpu memory free

__global__ void test()
{

	return;
}

// rgb->Ycbcr
__global__ void dev_convertColorSpace(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{
	int channels = 3;
	//dataSize /= 3;
	// blockid * wieVieleBl�ckeGesamt + (threadID*channels)
	int globalThreadId = (blockIdx.x * channels) * blockDim.x + (threadIdx.x*channels);

	// grid-stride loop
	for(int dataElement = globalThreadId; dataElement < (dataSize-(channels)); dataElement= dataElement+(gridDim.x * blockDim.x)*(channels))
	{
		unsigned char r = dev_data[dataElement + 0];
		unsigned char g = dev_data[dataElement +1];
		unsigned char b = dev_data[dataElement +2];
		//*r = 16+ (((*r << 6) + (*r << 1) + (*g << 7) + *g + (*b << 4) + (*b << 3) + *b) >> 8); // Y
		//*g= 128 + ((-((*r<<5)+(*r<<2)+(*r<<1))-((*g<<6)+(*g<<3)+(*g<<1))+(*b<<7)-(*b<<4))>>8); // Cb
		//*b = 128 + (((*r<<7)-(*r<<4)-((*g<<6)+(*g<<5)-(*g<<1))-((*b<<4)+(*b<<1)))>>8); // Cr
		//*(dev_dataResult + dataElement + 0) = *r;
		//*(dev_dataResult + dataElement + 1) = *g;
		//*(dev_dataResult + dataElement + 2) = *b;

		dev_dataResult[dataElement + 0] = 16 + (((r << 6) + (r << 1) + (g << 7) + g + (b << 4) + (b << 3) + b) >> 8); // Y
		dev_dataResult[dataElement + 1] = 128 + ((-((r << 5) + (r << 2) + (r << 1)) - ((g << 6) + (g << 3) + (g << 1)) + (b << 7) - (b << 4)) >> 8); // Cb
		dev_dataResult[dataElement + 2] = 128 + (((r << 7) - (r << 4) - ((g << 6) + (g << 5) - (g << 1)) - ((b << 4) + (b << 1))) >> 8); // Cr
	}
}


//// d_paddedImage: speicherallokierung mit ->  paddedIWidth * paddedIHeight * sizeof(float)
//const T* d_f,
//
//// paddedIWidth = iWidth + 2 * hFilterSize // hFilterSize = filterSize / 2 = eig radius des kernels
//const unsigned int paddedW,
//
//
//// paddedIHeight = iHeight + 2 * hFilterSize // hFilterSize = filterSize / 2 = eig radius des kernels
//const unsigned int paddedH,
//
//// radius of filter, sp�ter zusammengetragen mit filterSize=(2S+1)�(2S+1)(2S+1)�(2S+1).
//const int S,
//
//// d_filteringResult speicherallokierung mit // iWidth * iHeight * sizeof(float)
//T* d_h,
//
//// image widthe
//const unsigned int W,
//
////image height
//const unsigned int H )


__global__ void dev_applyGaussian(unsigned char* dev_data, unsigned char* dev_dataResult, double** filter, int dataSize, int imageHeight, int imageWidth, int filterHeight)
{

	int filterWidth = filterHeight;
	int newImageHeight = imageHeight - filterHeight + 1;
	int newImageWidth = imageWidth - filterHeight + 1;


		for (int i = 0; i < newImageWidth; i++) {
			for (int j = 0; j < newImageHeight; j++) {
				for (int h = i; h < i + filterWidth; h++) {
					for (int w = j; w < j + filterHeight; w++) {
						dev_dataResult[i* imageWidth + j] = dev_dataResult[i * imageWidth + j] + filter[h - i][w - j] * dev_data[h * imageWidth + w];
					}
				}
			}
		}

}


void doSmth()
{
	std::cout << "Hi from doSmth" << std::endl;
	test << < 1, 1 >> > ();
	return;
}



unsigned char * convertRGBToYCBCR(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	cout << dataSize << endl;
	unsigned char* dataResult = (unsigned char*) malloc(sizeof(unsigned char) * dataSize);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_convertColorSpace <<< gridDims, blockDims >>> (dev_data, dev_dataResult, dataSize);


	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);


	hipFree(&dev_data);
	hipFree(&dev_dataResult);



	return dataResult;
}



unsigned char* gaussianOneChannel(unsigned char * data, int dataSize, dim3 gridDims, dim3 blockDims, double** filter, int imageHeight, int imageWidth, int filterHeight)
{
	int newImageWidth = imageWidth - filterHeight + 1;
	int newImageHeight = imageHeight - filterHeight + 1;
	int dataSizeResultImage = newImageWidth * newImageHeight;
	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSizeResultImage);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;


	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize/3);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSizeResultImage);
	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize / 3, hipMemcpyHostToDevice);
	dev_applyGaussian << < gridDims, blockDims >> > (dev_data, dev_dataResult, filter, dataSize, imageHeight, imageWidth, filterHeight);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}


	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSizeResultImage, hipMemcpyDeviceToHost);
	hipFree(&dev_data);
	hipFree(&dev_dataResult);
	return dataResult;
}

double** createGaussianFilter(int width, int height, double sigma)
{
	double PI = 3.1415;

	double** kernel = (double**) malloc(height * sizeof(double *));

	for (int i = 0; i < height; i++) {
		kernel[i] = (double*) malloc(sizeof(double) * width);
	}


	double sum = 0.0;
	int a, b;

	for (a = 0; a < height; a++) {

		for (b = 0; b < width; b++) {
			cout << "hier ";
			double result = exp(-(a * a + b * b) / (2 * sigma * sigma)) / (2 * PI * sigma * sigma);
			cout << "result: " << result << endl;
			kernel[a][b] = result;
			sum += kernel[a][b];
		}
	}

	for (a = 0; a < height; a++) {

		for (b = 0; b < width; b++) {
			kernel[a][b] /= sum;
		}
	}

	cout << "sum: " << sum << endl;

	//for (a = 0; a < width; a++)
	//{
	//	for (b = 0; b < height; b++)
	//	{
	//		cout << (double) kernel[a][b] << " ";
	//	}
	//	cout << endl;
	//}

	return kernel;
}

// data: BGR-Sequence of the input channels of data
unsigned char** applyGaussianFilter(unsigned char** data, const int dataSize, dim3 gridDims, dim3 blockDims, const int channelsPara, int imageHeight, int imageWidth, int filterHeight, double sigma)
{
	cout << dataSize << endl;
	const int channels = channelsPara;

	const int sizeOfOneColorChannel = dataSize / 3;

	unsigned char** resultChannels1 = (unsigned char**)malloc(channels * sizeof(unsigned char*));

	double** filter = createGaussianFilter(filterHeight, filterHeight, sigma);

	for (int i = 0; i < channels; i++) {
		resultChannels1[i] = (unsigned char*) malloc(sizeof(unsigned char) * sizeOfOneColorChannel);
		resultChannels1[i] = gaussianOneChannel(data[i], dataSize, gridDims, blockDims, filter, imageHeight, imageWidth, filterHeight);
	}

	hipDeviceReset();

	return resultChannels1;
}




void cudaMain(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	doSmth();
	convertRGBToYCBCR(data, dataSize, gridDims, blockDims);
	//applyGaussianFilter(data, dataSize, gridDims, blockDims);
}

