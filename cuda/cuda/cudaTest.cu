#include "hip/hip_runtime.h"
#include "cudaTest.cuh"

// cuda procedure
//
// gpu memory allocation
// copy cpu->gpu
// calculation- / kernel phase
// copy gpu->cpu
// gpu memory free

__global__ void test()
{

	return;
}

// rgb->Ycbcr
__global__ void dev_convertColorSpace(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{
	int channels = 3;
	//dataSize /= 3;
	// blockid * wieVieleBl�ckeGesamt + (threadID*channels)
	int globalThreadId = (blockIdx.x * channels) * blockDim.x + (threadIdx.x*channels);

	// grid-stride loop
	for(int dataElement = globalThreadId; dataElement < (dataSize-(channels)); dataElement= dataElement+(gridDim.x * blockDim.x)*(channels))
	{
		unsigned char r = dev_data[dataElement + 0];
		unsigned char g = dev_data[dataElement +1];
		unsigned char b = dev_data[dataElement +2];
		//*r = 16+ (((*r << 6) + (*r << 1) + (*g << 7) + *g + (*b << 4) + (*b << 3) + *b) >> 8); // Y
		//*g= 128 + ((-((*r<<5)+(*r<<2)+(*r<<1))-((*g<<6)+(*g<<3)+(*g<<1))+(*b<<7)-(*b<<4))>>8); // Cb
		//*b = 128 + (((*r<<7)-(*r<<4)-((*g<<6)+(*g<<5)-(*g<<1))-((*b<<4)+(*b<<1)))>>8); // Cr
		//*(dev_dataResult + dataElement + 0) = *r;
		//*(dev_dataResult + dataElement + 1) = *g;
		//*(dev_dataResult + dataElement + 2) = *b;

		dev_dataResult[dataElement + 0] = 16 + (((r << 6) + (r << 1) + (g << 7) + g + (b << 4) + (b << 3) + b) >> 8); // Y
		dev_dataResult[dataElement + 1] = 128 + ((-((r << 5) + (r << 2) + (r << 1)) - ((g << 6) + (g << 3) + (g << 1)) + (b << 7) - (b << 4)) >> 8); // Cb
		dev_dataResult[dataElement + 2] = 128 + (((r << 7) - (r << 4) - ((g << 6) + (g << 5) - (g << 1)) - ((b << 4) + (b << 1))) >> 8); // Cr
	}
}

__global__ void dev_applyGaussian(unsigned char* dev_data, unsigned char* dev_dataResult, int dataSize)
{
	int channels = 3;
	for (size_t i = 0; i < dataSize; i++)
	{

	}

	*dev_dataResult = 1;



    //double stdv = 1.0;
    //double r, s = 2.0 * stdv * stdv;  // Assigning standard deviation to 1.0
    //double sum = 0.0;   // Initialization of sun for normalization
    //for (int x = -2; x <= 2; x++) // Loop to generate 5x5 kernel
    //{
    //    for(int y = -2; y <= 2; y++)
    //    {
    //        r = sqrt(x*x + y*y);
    //        gk[x + 2][y + 2] = (exp(-(r*r)/s))/(M_PI * s);
    //        sum += gk[x + 2][y + 2];
    //    }
    //}

    //for(int i = 0; i < 5; ++i) // Loop to normalize the kernel
    //    for(int j = 0; j < 5; ++j)
    //        gk[i][j] /= sum;

}


void doSmth()
{
	std::cout << "Hi from doSmth" << std::endl;
	test << < 1, 1 >> > ();
	return;
}

unsigned char * convertRGBToYCBCR(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	cout << dataSize << endl;
	unsigned char* dataResult = (unsigned char*) malloc(sizeof(unsigned char) * dataSize);

	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);

	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);

	dev_convertColorSpace <<< gridDims, blockDims >>> (dev_data, dev_dataResult, dataSize);

	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);



	hipFree(&dev_data);
	hipFree(&dev_dataResult);



	return dataResult;
}



unsigned char* gaussianOneChannel(unsigned char * data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	unsigned char* dataResult = (unsigned char*)malloc(sizeof(unsigned char) * dataSize);
	unsigned char* dev_data;
	unsigned char* dev_dataResult;
	hipMalloc(&dev_data, sizeof(unsigned char) * dataSize);
	hipMalloc(&dev_dataResult, sizeof(unsigned char) * dataSize);
	hipMemcpy(dev_data, data, sizeof(unsigned char) * dataSize, hipMemcpyHostToDevice);
	dev_applyGaussian << < gridDims, blockDims >> > (dev_data, dev_dataResult, dataSize);
	hipMemcpy(dataResult, dev_dataResult, sizeof(unsigned char) * dataSize, hipMemcpyDeviceToHost);
	hipFree(&dev_data);
	hipFree(&dev_dataResult);
	return dataResult;
}



unsigned char** applyGaussianFilter(unsigned char** data, const int dataSize, dim3 gridDims, dim3 blockDims, const int channelsPara)
{
	cout << dataSize << endl;
	const int channels = channelsPara;

	const int sizeOfOneColorChannel = dataSize / 3;

	unsigned char** resultChannels1 = (unsigned char**)malloc(channels * sizeof(unsigned char*));

	for (int i = 0; i < channels; i++) {
		resultChannels1[i] = (unsigned char*) malloc(sizeof(unsigned char) * sizeOfOneColorChannel);
		resultChannels1[i] = gaussianOneChannel(data[i], dataSize, gridDims, blockDims);

	}
	
	
	
	hipDeviceReset();



	return resultChannels1;
}




void cudaMain(unsigned char* data, int dataSize, dim3 gridDims, dim3 blockDims)
{
	doSmth();
	convertRGBToYCBCR(data, dataSize, gridDims, blockDims);
	//applyGaussianFilter(data, dataSize, gridDims, blockDims);
}

